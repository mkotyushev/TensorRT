#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "hardshrinkPlugin.h"
#include <hip/hip_fp16.h>


template <typename T_DATA>
__global__ void kernelHardshrink(
    float lambd,
    T_DATA* inputs
    T_DATA* outputs
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    T_DATA input = inputs[index];
    T_DATA lambdConverted = T_DATA(lambd);

    outputs[index] = ((input > lambd) || (input < -lambd)) ? input : T_DATA(0.0);
    __syncthreads();
}

template <typename T>
int inferenceHardshrink(
    int size,
    float lambd,
    T* inputs,
    T* outputs,
    hipStream_t stream)
{
    const int nThreads = 512;
    int nBlocks = (int)((float)size / nThreads) + 1;

    kernelHardshrink<<<nBlocks, nThreads, 0, stream>>>(lambd, inputs, outputs);

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                __FILE__, __LINE__, hipGetErrorString( err ) );
        return 1;
    }
    return 0;
}

int HardshrinkPlugin::enqueue(
    int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
{
    switch(iType){
        case DataType::kFLOAT:
            return inferenceAC(mBatchDim * batchSize, mLambd, (float*)inputs[0], (float*)outputs[0], stream);
        case DataType::kHALF:
            return inferenceAC(mBatchDim * batchSize, mLambd, (__half*)inputs[0], (__half*)outputs[0], stream);
    }
    return 1;
}
